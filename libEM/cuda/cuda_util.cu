
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_defs.h"
#include "cuda_emfft.h"

#include "emcudautil.h"

texture<float, 3, hipReadModeElementType> tex;

void hipBindTexture(texture<float, 3, hipReadModeElementType> &tex,hipArray *array) {
	tex.normalized = 0;
	tex.filterMode = hipFilterModeLinear;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.addressMode[2] = hipAddressModeClamp;
	
	CUDA_SAFE_CALL(hipBindTextureToArray(tex, array));
}


struct CudaEMDataArray {
	hipArray* array;
	const float* data;
	void* emdata_pointer;
};

void copy_array_data(CudaEMDataArray* to, CudaEMDataArray* from) {
	to->array = from->array;
	to->data = from->data;
	to->emdata_pointer = from->emdata_pointer;
}

void set_arry_data_null(CudaEMDataArray* p)
{
	p->array = 0;
	p->data = 0;
	p->emdata_pointer = 0;
}

const int max_cuda_arrays = 100;
int num_cuda_arrays = 0;
CudaEMDataArray cuda_arrays[max_cuda_arrays];

void init_cuda_emdata_arrays() {
	for(int i = 0; i < max_cuda_arrays; ++i ) {
		CudaEMDataArray c =  { 0, 0 };
		cuda_arrays[i] = c;
	}
}

int get_cuda_array_handle(const float * data,const int nx, const int ny, const int nz, void* emdata_pointer) {
	
	for(int i = 0; i < num_cuda_arrays; ++i ) {
		if (cuda_arrays[i].data == data ) {
			//printf("Found that cuda arrary\n");
			return i;
		}
	}
	
	//printf("Making a new cuda array\n");
	// If we make it here then it doesn't exist
	cuda_arrays[num_cuda_arrays].data = data;
	cuda_arrays[num_cuda_arrays].emdata_pointer = emdata_pointer;
	hipExtent VS = make_hipExtent(nx,ny,nz);
	
	hipArray *array = 0;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&array, &channelDesc, VS);
	
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void*)data, VS.width*sizeof(float), VS.width, VS.height);
	copyParams.dstArray = array;
	copyParams.extent   = VS;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	
	//hipBindTexture(tex,array);
	cuda_arrays[num_cuda_arrays].array = array;
	cuda_arrays[num_cuda_arrays].emdata_pointer = emdata_pointer;
	num_cuda_arrays++;
	return num_cuda_arrays-1;
}

int delete_cuda_array(const int idx) {
	CUDA_SAFE_CALL(hipFree(cuda_arrays[idx].array));
	cuda_arrays[idx].array = 0;
	
	for (int i = idx; i < num_cuda_arrays-1; ++i ) {
		CudaEMDataArray* to = &cuda_arrays[idx];
		CudaEMDataArray* from = &cuda_arrays[idx+1];
		copy_array_data(to,from);
		set_emdata_cuda_array_handle(idx,to->emdata_pointer);
	}
	set_arry_data_null(&cuda_arrays[num_cuda_arrays-1]);
	num_cuda_arrays--;
	
	return 0;
}

int delete_cuda_memory(float*p) {
	CUDA_SAFE_CALL(hipFree(p));
	p = 0;
	return 0;
}

void bind_cuda_texture(const int idx) {
	//printf("Binding texture\n");
	CUDA_SAFE_CALL(hipBindTexture(tex,cuda_arrays[idx].array));
	//printf("Done bind\n");
}

void device_init() {
	static bool init = true;
	
	if (init) {
		int deviceCount;
		CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
		printf("%d CUDA devices detected\n",deviceCount);
		if (deviceCount == 0) exit(1);
		
		hipDeviceProp_t deviceProp;
		CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
		if (deviceProp.major < 1) exit(2);
		
		CUDA_SAFE_CALL(hipSetDevice(0));
		init_cuda_emdata_arrays();
		init_cuda_emfft_cache();
		init = false; //Force init everytikme
	}
}

void* cuda_malloc(const size_t size)
{
	device_init();
	void *mem=0;
	hipHostMalloc((void **)&mem, size);
	return mem;
}

void cuda_free(void* mem)
{
	device_init();
	hipHostFree(mem);
}
void cuda_memset(void* mem,int value, size_t size) {
	device_init();
	hipMemset(mem,value,size);
}

void cuda_memcpy(void* dst, const void* const src, size_t count) {
	device_init();
	hipMemcpy(dst,src,count,hipMemcpyHostToHost);
// 	hipStream_t stream;
// 	hipStreamCreate(&stream);
// 	hipMemcpyAsync(dst,src,count,hipMemcpyHostToHost,stream);
// 	hipStreamSynchronize(stream);
// 	hipStreamDestroy(stream);
}

