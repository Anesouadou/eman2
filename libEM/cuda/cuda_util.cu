
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_defs.h"
#include "cuda_emfft.h"

#include "emcudautil.h"

texture<float, 3, hipReadModeElementType> tex;

void hipBindTexture(texture<float, 3, hipReadModeElementType> &tex,hipArray *array) {
	tex.normalized = 0;
	tex.filterMode = hipFilterModeLinear;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.addressMode[2] = hipAddressModeClamp;
	
	CUDA_SAFE_CALL(hipBindTextureToArray(tex, array));
}


struct CudaEMDataArray {
	hipArray* array;
	const float* data; /*This one may be unecessary*/
	void* emdata_pointer;
};

void copy_array_data(CudaEMDataArray* to, CudaEMDataArray* from) {
	to->array = from->array;
	to->data = from->data;
	to->emdata_pointer = from->emdata_pointer;
}

void set_array_data_null(CudaEMDataArray* p)
{
	p->array = 0;
	p->data = 0;
	p->emdata_pointer = 0;
}

const int max_cuda_arrays = 2;
int num_cuda_arrays = 0;
CudaEMDataArray cuda_arrays[max_cuda_arrays];

void init_cuda_emdata_arrays() {
	for(int i = 0; i < max_cuda_arrays; ++i ) {
		CudaEMDataArray c =  { 0, 0 };
		cuda_arrays[i] = c;
	}
}


void debug_arrays()
{
	printf("debug\n");
	for(int i=0; i < max_cuda_arrays; ++i) 
	{
		CudaEMDataArray t = cuda_arrays[i];
		printf("%d %d %d\n",t.array,t.data,t.emdata_pointer);
	}
	printf("end debug\n");
}

int make_cuda_array_space_0_free() {
	//printf("Freeing space 0\n");
	//debug_arrays();
	int n = num_cuda_arrays-1;
	set_emdata_cuda_array_handle(-1,cuda_arrays[n].emdata_pointer);
	CUDA_SAFE_CALL(hipFree(cuda_arrays[n].array));
	cuda_arrays[n].array = 0;
	
	for (int i = 0; i < num_cuda_arrays-1; ++i ) {
		CudaEMDataArray* to = &cuda_arrays[i+1];
		CudaEMDataArray* from = &cuda_arrays[i];
		copy_array_data(to,from);
		set_emdata_cuda_array_handle(i+1,to->emdata_pointer);
	}
	set_array_data_null(&cuda_arrays[0]);
	
	//debug_arrays();
	return 0;
}


int get_cuda_array_handle(const float * data,const int nx, const int ny, const int nz, void* emdata_pointer) {
	
	//printf("Get cuda array %d\n", emdata_pointer);
	//debug_arrays();
	for(int i = 0; i < num_cuda_arrays; ++i ) {
		if (cuda_arrays[i].emdata_pointer == emdata_pointer ) {
			//printf("Found that cuda arrary\n");
			return i;
		}
	}
	int idx = num_cuda_arrays;
	if (num_cuda_arrays == max_cuda_arrays) {
		make_cuda_array_space_0_free();
		idx = 0;
	}
	//printf("Making a new cuda array\n");
	// If we make it here then it doesn't exist
	cuda_arrays[idx].data = data;
	cuda_arrays[idx].emdata_pointer = emdata_pointer;
	hipExtent VS = make_hipExtent(nx,ny,nz);
	
	hipArray *array = 0;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&array, &channelDesc, VS);
	
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void*)data, VS.width*sizeof(float), VS.width, VS.height);
	copyParams.dstArray = array;
	copyParams.extent   = VS;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	
	cuda_arrays[idx].array = array;
	if (num_cuda_arrays != max_cuda_arrays) num_cuda_arrays++;
	return idx;
}

int delete_cuda_array(const int idx) {
	//printf("Deleting a cuda array\n");
	CUDA_SAFE_CALL(hipFree(cuda_arrays[idx].array));
	cuda_arrays[idx].array = 0;
	
	for (int i = idx; i < num_cuda_arrays; ++i ) {
		CudaEMDataArray* to = &cuda_arrays[i];
		CudaEMDataArray* from = &cuda_arrays[i+1];
		copy_array_data(to,from);
		set_emdata_cuda_array_handle(i,to->emdata_pointer);
	}
	set_array_data_null(&cuda_arrays[num_cuda_arrays-1]);
	num_cuda_arrays--;
	
	return 0;
}

// highly specialized, use cautiously


int delete_cuda_memory(float*p) {
	CUDA_SAFE_CALL(hipFree(p));
	p = 0;
	return 0;
}

void bind_cuda_texture(const int idx) {
	//printf("Binding texture\n");
	CUDA_SAFE_CALL(hipBindTexture(tex,cuda_arrays[idx].array));
	//printf("Done bind\n");
}

void device_init() {
	static bool init = true;
	
	if (init) {
		int deviceCount;
		CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
		printf("%d CUDA devices detected\n",deviceCount);
		if (deviceCount == 0) exit(1);
		
		hipDeviceProp_t deviceProp;
		CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
		if (deviceProp.major < 1) exit(2);
		
		CUDA_SAFE_CALL(hipSetDevice(0));
		init_cuda_emdata_arrays();
		init_cuda_emfft_cache();
		init = false; //Force init everytikme
	}
}

void* cuda_malloc(const size_t size)
{
	device_init();
	void *mem=0;
	hipHostMalloc((void **)&mem, size);
	return mem;
}

void cuda_free(void* mem)
{
	device_init();
	hipHostFree(mem);
}
void cuda_memset(void* mem,int value, size_t size) {
	device_init();
	hipMemset(mem,value,size);
}

void cuda_memcpy(void* dst, const void* const src, size_t count) {
	device_init();
	hipMemcpy(dst,src,count,hipMemcpyHostToHost);
// 	hipStream_t stream;
// 	hipStreamCreate(&stream);
// 	hipMemcpyAsync(dst,src,count,hipMemcpyHostToHost,stream);
// 	hipStreamSynchronize(stream);
// 	hipStreamDestroy(stream);
}

