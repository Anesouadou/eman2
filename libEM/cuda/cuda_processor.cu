#include "hip/hip_runtime.h"


#include "cuda_util.h"
#include <stdio.h>

// Global texture
extern texture<float, 3, hipReadModeElementType> tex;
extern texture<float, 2, hipReadModeElementType> tex2d;

typedef unsigned int uint;

#ifdef WIN32
	#define M_PI 3.14159265358979323846f
#endif	//WIN32

__global__ void mult_kernel(float *data,const float scale,const int num_threads)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	data[x+y*num_threads] *= scale;
}

void emdata_processor_mult( EMDataForCuda* cuda_data, const float& mult) {
	
	int max_threads = 512;

	int num_calcs = cuda_data->nx*cuda_data->ny*cuda_data->nz;
	
	int grid_y = num_calcs/max_threads;
	int res_y = num_calcs - (grid_y*max_threads);
	
	if ( grid_y > 0 ) {
		const dim3 blockSize(max_threads,1, 1);
		const dim3 gridSize(grid_y,1,1);
		mult_kernel<<<gridSize,blockSize>>>(cuda_data->data,mult,max_threads);
	}
	
	if ( res_y > 0 ) {
		const dim3 blockSize(res_y,1, 1);
		const dim3 gridSize(1,1,1);
		mult_kernel<<<gridSize,blockSize>>>(cuda_data->data+grid_y*max_threads,mult,0);
	}

	hipDeviceSynchronize();	
}

__global__ void add_kernel(float *data,const float add,const int num_threads)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	data[x+y*num_threads] += add;
}

void emdata_processor_add( EMDataForCuda* cuda_data, const float& add) {
	
	int max_threads = 512;

	int num_calcs = cuda_data->nx*cuda_data->ny*cuda_data->nz;
	
	int grid_y = num_calcs/max_threads;
	int res_y = num_calcs - (grid_y*max_threads);
	
	if ( grid_y > 0 ) {
		const dim3 blockSize(max_threads,1, 1);
		const dim3 gridSize(grid_y,1,1);
		add_kernel<<<gridSize,blockSize>>>(cuda_data->data,add,max_threads);
	}
	
	if ( res_y > 0 ) {
		const dim3 blockSize(res_y,1, 1);
		const dim3 gridSize(1,1,1);
		add_kernel<<<gridSize,blockSize>>>(cuda_data->data+grid_y*max_threads,add,0);
	}

	hipDeviceSynchronize();	
}

__global__ void assignment_kernel(float *data,const float value,const int num_threads)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	data[x+y*num_threads] = value;
}

void emdata_processor_to_value( EMDataForCuda* cuda_data, const float& value) {
	
	int max_threads = 512;

	int num_calcs = cuda_data->nx*cuda_data->ny*cuda_data->nz;
	
	int grid_y = num_calcs/max_threads;
	int res_y = num_calcs - (grid_y*max_threads);
	
	if ( grid_y > 0 ) {
		const dim3 blockSize(max_threads,1, 1);
		const dim3 gridSize(grid_y,1,1);
		assignment_kernel<<<gridSize,blockSize>>>(cuda_data->data,value,max_threads);
	}
	
	if ( res_y > 0 ) {
		const dim3 blockSize(res_y,1, 1);
		const dim3 gridSize(1,1,1);
		assignment_kernel<<<gridSize,blockSize>>>(cuda_data->data+grid_y*max_threads,value,0);
	}

	hipDeviceSynchronize();	
}


__global__ void phaseorigin_to_center_fourier(float* data, const int num_threads, const int nx, const int ny, const int nz, const int offset)
{
	const uint x=threadIdx.x;
	const uint y=blockIdx.x;
	
	uint idx = x+y*num_threads+offset;
	const uint nxy = (nx/4)*ny;
	uint zz = idx/(nxy);
	uint yy = (idx-zz*nxy)/(nx/4);

	const uint xx = 4*(idx%(nx/4));
	
	const uint rnxy = nx*ny;
	const uint xoff = ((yy+zz)%2==0?2:0);
	const uint didx = zz*rnxy+yy*nx+xx+xoff;
	data[didx] *= -1;
	data[didx+1] *= -1;
}

void emdata_phaseorigin_to_center_fourier(const EMDataForCuda* cuda_data) {
	int nx = cuda_data->nx;
	int ny = cuda_data->ny;
	int nz = cuda_data->nz;
	float* data = cuda_data->data;
	
	if ( nx%2==0 && (ny%2==0 || ny==1 ) && (nz%2==0 || nz==1 ) ) {
		int max_threads = 512;
		
		int num_calcs = nz*ny*(nx/4);
			
		int grid_y = num_calcs/(max_threads);
		int res_y = num_calcs - grid_y*max_threads;

		//int odd_offset=0;
		//if (((ny/2)%2)+((nz/2)%2)==1) odd_offset=1;
		if (grid_y > 0) {
			const dim3 blockSize(max_threads,1, 1);
			const dim3 gridSize(grid_y,1,1);
			phaseorigin_to_center_fourier<<<gridSize,blockSize>>>(data,max_threads,nx,ny,nz,0);
		}
		
		if (res_y > 0) {
			const dim3 blockSize(res_y,1, 1);
			const dim3 gridSize(1,1,1);
			phaseorigin_to_center_fourier<<<gridSize,blockSize>>>(data,max_threads,nx,ny,nz,grid_y*max_threads);
		}
		hipDeviceSynchronize();
	} else {
		throw;
	}
}

__global__ void correlation_kernel(float *ldata, float* rdata,const int num_threads)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	const uint idx = 2*x + y*num_threads;
	const uint idxp1 = idx+1;
	
	const float v1 = ldata[idx];
	const float v2 = ldata[idxp1];
	const float u1 = rdata[idx];
	const float u2 = rdata[idxp1];
	
	ldata[idx] = v1*u1 + v2*u2;
	ldata[idxp1] = v1*u2 - v2*u1;
}

__global__ void auto_correlation_kernel(float *ldata, float* rdata,const int num_threads)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	const uint idx = 2*x + y*num_threads;
	const uint idxp1 = idx+1;
	
	const float v1 = ldata[idx];
	const float v2 = ldata[idxp1];
	const float u1 = rdata[idx];
	const float u2 = rdata[idxp1];
	
	ldata[idx] = v1*u1 + v2*u2;
	ldata[idxp1] = 0;
}


__global__ void correlation_kernel_texture_2D(float *ldata,const int num_threads,const int xsize,const int offset)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	const uint idx = 2*x + y*num_threads+offset;
	const uint idxp1 = idx+1;
	
	const uint tx = idx % xsize;
	const uint ty = idx / xsize;
	
	const float v1 = ldata[idx];
	const float v2 = ldata[idxp1];
	const float u1 = tex2D(tex2d,tx,ty);
	const float u2 =  tex2D(tex2d,tx+1,ty);
	
	ldata[idx] = v1*u1 + v2*u2;
	ldata[idxp1] = v1*u2 - v2*u1;
}


__global__ void correlation_kernel_texture_3D(float *ldata,const int num_threads, const int xsize, const int xysize, const int offset)
{

	const uint x=threadIdx.x;
	const uint y=blockIdx.x;

	const uint idx = 2*x + y*num_threads + offset;
	const uint idxp1 = idx+1;
	
	const uint tx = idx % xsize;
	const uint tz = idx / xysize;
	const uint ty = (idx - tz*xysize)/xsize;
	
	const float v1 = ldata[idx];
	const float v2 = ldata[idxp1];
	const float u1 = tex3D(tex,tx,ty,tz);
	const float u2 = tex3D(tex,tx+1,ty,tz);
	
	ldata[idx] = v1*u1 + v2*u2;
	ldata[idxp1] = v1*u2 - v2*u1;
}

void emdata_processor_correlation_texture( const EMDataForCuda* cuda_data, const int center ) {
	int max_threads = 512; // I halve the threads because each kernel access memory in two locations

	int num_calcs = cuda_data->nx*cuda_data->ny*cuda_data->nz;
	
	int grid_y = num_calcs/(2*max_threads);
	int res_y = (num_calcs - (2*grid_y*max_threads))/2;
	
// 	printf("Grid %d, Res %d, dims %d %d %d\n",grid_y,res_y,cuda_data->nx,cuda_data->ny,cuda_data->nz);
	
	if ( grid_y > 0 ) {
		const dim3 blockSize(max_threads,1, 1);
		const dim3 gridSize(grid_y,1,1);
		if (cuda_data->nz == 1) {
			correlation_kernel_texture_2D<<<gridSize,blockSize>>>(cuda_data->data,2*max_threads,cuda_data->nx,0);
		} else {
			correlation_kernel_texture_3D<<<gridSize,blockSize>>>(cuda_data->data,2*max_threads,cuda_data->nx,cuda_data->nx*cuda_data->ny,0);
		}
	}
// 	res_y = 0;
	if ( res_y > 0 ) {
		const dim3 blockSize(res_y,1,1);
		const dim3 gridSize(1,1,1);
		int inc = 2*grid_y*max_threads;
// 		printf("Res %d, inc %d\n",res_y,inc);
		if (cuda_data->nz == 1) {
			correlation_kernel_texture_2D<<<gridSize,blockSize>>>(cuda_data->data,0,cuda_data->nx,inc);
		} else {
			correlation_kernel_texture_3D<<<gridSize,blockSize>>>(cuda_data->data,0,cuda_data->nx,cuda_data->nx*cuda_data->ny,inc);
		}
	}
	
	hipDeviceSynchronize();
	if (center) {
		emdata_phaseorigin_to_center_fourier(cuda_data);
	}
}


void emdata_processor_correlation( const EMDataForCuda* left, const EMDataForCuda* right, const int center) {
	int max_threads = 512;

	int num_calcs = left->nx*left->ny*left->nz;
	
	int grid_y = num_calcs/(2*max_threads);
	int res_y = (num_calcs - (2*grid_y*max_threads))/2;
	
	//printf("Grid y %d, res %d, dims %d %d %d\n", grid_y,res_y,left->nx,left->ny,left->nz);
	
	if ( grid_y > 0 ) {
		const dim3 blockSize(max_threads,1, 1);
		const dim3 gridSize(grid_y,1,1);
		if (left->data != right->data) {
			correlation_kernel<<<gridSize,blockSize>>>(left->data,right->data,2*max_threads);
		} else {
			auto_correlation_kernel<<<gridSize,blockSize>>>(left->data,right->data,2*max_threads);
		}
	}
	
	if ( res_y > 0 ) {
		const dim3 blockSize(res_y,1, 1);
		const dim3 gridSize(1,1,1);
		int inc = 2*grid_y*max_threads;
		if (left->data != right->data) {
			correlation_kernel<<<gridSize,blockSize>>>(left->data+inc,right->data+inc,0);
		} else {
			auto_correlation_kernel<<<gridSize,blockSize>>>(left->data+inc,right->data+inc,0);
		}
	}
	hipDeviceSynchronize();
	
	if (center) {
		emdata_phaseorigin_to_center_fourier(left);
	}
}

__global__ void unwrap_kernel(float* dptr, const int num_threads, const int r1, const float p, const int nx, const int ny, const int nxp, const int dx,const int dy,const int weight_radial,const int offset) {
	const uint x=threadIdx.x;
	const uint y=blockIdx.x;
	
	const uint idx = x + y*num_threads+offset;
	
	const uint tx = idx % nxp;
	const uint ty = idx / nxp;
	
	float ang = tx * M_PI * p;
	float si = sinf(ang);
	float co = cosf(ang);

	float ypr1 = ty + r1;
	float xx = ypr1 * co + nx / 2 + dx;
	float yy = ypr1 * si + ny / 2 + dy;
	if ( weight_radial ) dptr[idx] = tex2D(tex2d,xx+0.5,yy+0.5)*ypr1;
	else dptr[idx] = tex2D(tex2d,xx+0.5,yy+0.5);
}

	
EMDataForCuda* emdata_unwrap(int r1, int r2, int xs, int num_pi, int dx, int dy, int weight_radial, int nx, int ny) {	
	
	float* dptr;
	int n = xs*(r2-r1);
	hipError_t error = hipMalloc((void**)&dptr,n*sizeof(float));
	if ( error != hipSuccess ) {
		const char* s = hipGetErrorString(error);
		printf("Cuda malloc failed in emdata_unwrap: %s\n",s);
		throw;
	}
	
	int max_threads = 512;
	int num_calcs = n;
	
	int grid_y = num_calcs/(max_threads);
	int res_y = num_calcs - grid_y*max_threads;
	
	//printf("Grid %d, res %d, n %d, p %f \n",grid_y,res_y,n, p/xs);
	
	if ( grid_y > 0 ) {
		const dim3 blockSize(max_threads,1, 1);
		const dim3 gridSize(grid_y,1,1);
		unwrap_kernel<<<gridSize,blockSize>>>(dptr,max_threads,r1,(float) num_pi/ (float)xs, nx,ny,xs,dx,dy,weight_radial,0);	
	}
	
	if ( res_y > 0 ) {
		const dim3 blockSize(res_y,1, 1);
		const dim3 gridSize(1,1,1);
		unwrap_kernel<<<gridSize,blockSize>>>(dptr,max_threads,r1, (float) num_pi/ (float)xs, nx,ny,xs,dx,dy,weight_radial,grid_y*max_threads);	
	}
	
	EMDataForCuda* tmp = (EMDataForCuda*) malloc( sizeof(EMDataForCuda) );
	tmp->data = dptr;
	tmp->nx = xs;
	tmp->ny = r2-r1;
	tmp->nz = 1;
	return tmp;
}



__global__ void swap_bot_left_top_right(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
	const uint x=threadIdx.x;
	const uint y=blockIdx.x;
	
	const uint gpu_idx = x+y*num_threads+offset; 
	const uint c = gpu_idx % (nx/2);
	const uint r = gpu_idx / (nx/2);
	
	const uint idx1 = r*nx + c;
	const uint idx2 = (r+ny/2+yodd)*nx + c + nx/2+xodd;
	float tmp = data[idx1];
	data[idx1] = data[idx2];
	data[idx2] = tmp;
}

__global__ void swap_top_left_bot_right(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
	const uint x=threadIdx.x;
	const uint y=blockIdx.x;
	
	const uint gpu_idx = x+y*num_threads+offset;
	const uint c = gpu_idx % (nx/2);
	const uint r = gpu_idx / (nx/2) + ny/2+yodd;
	
	const uint idx1 = r*nx + c;
	const uint idx2 = (r-ny/2-yodd)*nx + c + nx/2+xodd;
	float tmp = data[idx1];
	data[idx1] = data[idx2];
	data[idx2] = tmp;
}

__global__ void swap_middle_row(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
	const uint x=threadIdx.x;
	const uint y=blockIdx.x;
	
	const uint c = x+y*num_threads+offset;
	int r = ny/2;
	int idx1 = r*nx + c;
	int idx2 = r*nx + c + nx/2+ xodd;
	float tmp = data[idx1];
	data[idx1] = data[idx2];
	data[idx2] = tmp;
}

// Iterate along the central column, swapping values where appropriate
__global__ void swap_middle_column(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
	const uint x=threadIdx.x;
	const uint y=blockIdx.x;
	
	const uint r = x+y*num_threads+offset;
	int c = nx/2;
	int idx1 = r*nx + c;
	int idx2 = (r+ny/2+yodd)*nx + c;
	float tmp = data[idx1];
	data[idx1] = data[idx2];
	data[idx2] = tmp;
}

void swap_central_slices_180(EMDataForCuda* cuda_data)
{
	int nx = cuda_data->nx;
	int ny = cuda_data->ny;
	int nz = cuda_data->nz;

	int xodd = (nx % 2) == 1;
	int yodd = (ny % 2) == 1;
	//int zodd = (nz % 2) == 1;
	
	//int nxy = nx * ny;
	float *data = cuda_data->data;

	if ( ny == 1 && nz == 1 ){
		throw;
	}
	else if ( nz == 1 ) {
		if ( yodd ) {
			// Iterate along middle row, swapping values where appropriate
			
			int max_threads = 512;
			int num_calcs = nx/2;
				
			int grid_y = num_calcs/(max_threads);
			int res_y = num_calcs - grid_y*max_threads;
			
			if (grid_y > 0) {
				const dim3 blockSize(max_threads,1, 1);
				const dim3 gridSize(grid_y,1,1);
				swap_middle_row<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,0);
			}
			
			if (res_y) {
				const dim3 blockSize(res_y,1, 1);
				const dim3 gridSize(1,1,1);
				swap_middle_row<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,grid_y*max_threads);
			}
		}

		if ( xodd )	{
			// Iterate along the central column, swapping values where appropriate
			int max_threads = 512;
			int num_calcs = ny/2;
				
			int grid_y = num_calcs/(max_threads);
			int res_y = num_calcs - grid_y*max_threads;
			
			if (grid_y > 0) {
				const dim3 blockSize(max_threads,1, 1);
				const dim3 gridSize(grid_y,1,1);
				swap_middle_column<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,0);
			}
			
			if (res_y) {
				const dim3 blockSize(res_y,1, 1);
				const dim3 gridSize(1,1,1);
				swap_middle_column<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,grid_y*max_threads);
			}
			
		}
	}
	else // nx && ny && nz are greater than 1
	{
		throw;
	}
}

void swap_corners_180(EMDataForCuda* cuda_data)
{
	int nx = cuda_data->nx;
	int ny = cuda_data->ny;
	int nz = cuda_data->nz;

	int xodd = (nx % 2) == 1;
	int yodd = (ny % 2) == 1;
	//int zodd = (nz % 2) == 1;

	//int nxy = nx * ny;

	float *data = cuda_data->data;

	if ( ny == 1 && nz == 1 ){
		throw;
	}
	else if ( nz == 1 ) {
		int max_threads = 512;
		int num_calcs = ny/2*nx/2;
			
		int grid_y = num_calcs/(max_threads);
		int res_y = num_calcs - grid_y*max_threads;
		
		//printf("Grid %d, res %d, n %d\n",grid_y,res_y,num_calcs );
		// Swap bottom left and top right
		if (grid_y > 0) {
			const dim3 blockSize(max_threads,1, 1);
			const dim3 gridSize(grid_y,1,1);
			swap_bot_left_top_right<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,0);
		}
		
		if (res_y) {
			const dim3 blockSize(res_y,1, 1);
			const dim3 gridSize(1,1,1);
			swap_bot_left_top_right<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,grid_y*max_threads);
		}
		
		num_calcs = (ny-ny/2+yodd)*nx/2;
		//printf("Grid %d, res %d, n %d\n",grid_y,res_y,num_calcs );
					
		grid_y = num_calcs/(max_threads);
		res_y = num_calcs - grid_y*max_threads;
		// Swap the top left and bottom right corners
		if (grid_y > 0) {
			const dim3 blockSize(max_threads,1, 1);
			const dim3 gridSize(grid_y,1,1);
			swap_top_left_bot_right<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,0);
		}
		
		if (res_y) {
			const dim3 blockSize(res_y,1, 1);
			const dim3 gridSize(1,1,1);
			swap_top_left_bot_right<<<gridSize,blockSize>>>(data,max_threads,nx,ny,xodd,yodd,grid_y*max_threads);

		}
	}
	else // nx && ny && nz are greater than 1
	{
		throw;
	}
}

__global__ void middle_to_right(float* data, const int nx, const int ny)
{
	float tmp;
	for ( int r  = 0; r < ny; ++r ) {
		float last_val = data[r*nx+nx/2];
		for ( int c = nx-1; c >=  nx/2; --c ){
			int idx = r*nx+c;
			tmp = data[idx];
			data[idx] = last_val;
			last_val = tmp;
		}
	}
}

__global__ void middle_to_top(float* data, const int nx, const int ny)
{
	float tmp;
	for ( int c = 0; c < nx; ++c ) {
		// Get the value in the top row
		float last_val = data[ny/2*nx + c];
		for ( int r = ny-1; r >= ny/2; --r ){
			int idx = r*nx+c;
			tmp = data[idx];
			data[idx] = last_val;
			last_val = tmp;
		}
	}
}


void emdata_phaseorigin_to_center(EMDataForCuda* cuda_data) {
	int xodd = (cuda_data->nx % 2) == 1;
	int yodd = (cuda_data->ny % 2) == 1;
	//int zodd = (cuda_data->nz % 2) == 1;

	//int nxy = nx * ny;
	if ( cuda_data->nz == 1 && cuda_data->ny > 1 ){
		// The order in which these operations occur literally undoes what the
		// PhaseToCornerProcessor did to the image.
		// First, the corners sections of the image are swapped appropriately
		swap_corners_180(cuda_data);
		// Second, central pixel lines are swapped
		swap_central_slices_180(cuda_data);

		// Third, appropriate sections of the image are cyclically shifted by one pixel
		if (xodd) {
			// Transfer the middle column to the far right
			// Shift all from the far right to (but not including the) middle one to the left
			middle_to_right<<<1,1>>>(cuda_data->data,cuda_data->nx,cuda_data->ny);
		}
		if (yodd) {
			// Tranfer the middle row to the top,
			// shifting all pixels from the top row down one, until  but not including the) middle
			middle_to_top<<<1,1>>>(cuda_data->data,cuda_data->nx,cuda_data->ny);
		}
		hipDeviceSynchronize();
	} else {
		throw;
	}
}



